#include "hip/hip_runtime.h"
﻿/*---------------------------------------------------------------------------------*/
/*  COVID GPU - GPU accelerated functions for agent-based modelling -              */
/*                                                                                 */
/*  COVID GPU - version 1.0.0 has been created by                                  */
/*                 Khalil Al Handawi           - McGill University                 */
/*                                                                                 */
/*  The copyright of NOMAD - version 3.9.1 is owned by                             */
/*                 Khalil Al Handawi           - McGill University                 */
/*                                                                                 */
/*                                                                                 */
/*  Contact information:                                                           */
/*    McGill University - Systems Optimization Lab (SOL)                           */
/*    Macdonald Engineering Building, 817 Sherbrooke Street West,                  */
/*    Montreal (Quebec) H3A 0C3 Canada                                             */
/*    e-mail: khalil.alhandawi@mail.mcgill.ca                                      */
/*    phone : 1-514-398-2343                                                       */
/*                                                                                 */
/*  This program is free software: you can redistribute it and/or modify it        */
/*  under the terms of the GNU Lesser General Public License as published by       */
/*  the Free Software Foundation, either version 3 of the License, or (at your     */
/*  option) any later version.                                                     */
/*                                                                                 */
/*  This program is distributed in the hope that it will be useful, but WITHOUT    */
/*  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or          */
/*  FITNESS FOR A PARTICULAR PURPOSE. See the GNU Lesser General Public License    */
/*  for more details.                                                              */
/*                                                                                 */
/*  You should have received a copy of the GNU Lesser General Public License       */
/*  along with this program. If not, see <http://www.gnu.org/licenses/>.           */
/*                                                                                 */
/*---------------------------------------------------------------------------------*/

/**
 \file   CUDA_functions.cu
 \brief  GPU accelerate matrix functions (implementation)
 \author Khalil Al Handawi
 \date   2021-01-11
 \see    CUDA_functions.cuh
 */
#include <thrust/host_vector.h>
#include  "hip/hip_runtime.h"
#include ""
#include <hipblas.h>

#include <iostream>
#include <stdio.h>
#include <assert.h>
#include <Eigen/Core>

#include "CUDA_functions.h"
#include "kernels.cuh"
#include "Utilities.cuh"

using namespace std;

/*-----------------------------------------------------------*/
/*             CUBLAS ERROR MESSAGES ENUMERATOR              */
/*-----------------------------------------------------------*/
static const char *_cublasGetErrorEnum(hipblasStatus_t error)
{
	switch (error)
	{
	case HIPBLAS_STATUS_SUCCESS:
		return "HIPBLAS_STATUS_SUCCESS";

	case HIPBLAS_STATUS_NOT_INITIALIZED:
		return "HIPBLAS_STATUS_NOT_INITIALIZED";

	case HIPBLAS_STATUS_ALLOC_FAILED:
		return "HIPBLAS_STATUS_ALLOC_FAILED";

	case HIPBLAS_STATUS_INVALID_VALUE:
		return "HIPBLAS_STATUS_INVALID_VALUE";

	case HIPBLAS_STATUS_ARCH_MISMATCH:
		return "HIPBLAS_STATUS_ARCH_MISMATCH";

	case HIPBLAS_STATUS_MAPPING_ERROR:
		return "HIPBLAS_STATUS_MAPPING_ERROR";

	case HIPBLAS_STATUS_EXECUTION_FAILED:
		return "HIPBLAS_STATUS_EXECUTION_FAILED";

	case HIPBLAS_STATUS_INTERNAL_ERROR:
		return "HIPBLAS_STATUS_INTERNAL_ERROR";

	case HIPBLAS_STATUS_NOT_SUPPORTED:
		return "HIPBLAS_STATUS_NOT_SUPPORTED";

	case HIPBLAS_STATUS_UNKNOWN:
		return "HIPBLAS_STATUS_UNKNOWN";
	}

	return "<unknown>";
}

/*-----------------------------------------------------------*/
/*                   CUBLAS ERROR CHECKING                   */
/*-----------------------------------------------------------*/
inline void __cublasSafeCall(hipblasStatus_t err, const char *file, const int line)
{
	if (HIPBLAS_STATUS_SUCCESS != err) {
		fprintf(stderr, "CUBLAS error in file '%s', line %d, error: %s\nterminating!\n", __FILE__, __LINE__, \
			_cublasGetErrorEnum(err)); \
			assert(0); \
	}
}

/*-----------------------------------------------------------*/
/*               CUBLAS ERROR CHECKING (macro)               */
/*-----------------------------------------------------------*/
#define cublascheck(ans) { __cublasSafeCall((ans), __FILE__, __LINE__); }

/*-----------------------------------------------------------*/
/*                    CUDA ERROR CHECKING                    */
/*-----------------------------------------------------------*/
inline void _check(hipError_t code, char *file, int line)
{
	if (code != hipSuccess) {
		fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(code), file, line);
		exit(code);
	}
}

/*-----------------------------------------------------------*/
/*                CUDA ERROR CHECKING (macro)                */
/*-----------------------------------------------------------*/
#define check(ans) { _check((ans), __FILE__, __LINE__); }

/*-----------------------------------------------------------*/
/*              Repulsive force evaluation (GPU)             */
/*-----------------------------------------------------------*/
DLL_API void pairwise_gpu(Eigen::ArrayXf *force_x, Eigen::ArrayXf *force_y, Eigen::ArrayXf atoms_x, 
	Eigen::ArrayXf atoms_y, float SD_factor, int threads_per_block)
{

	const int N(atoms_x.rows()); // number of elements

	//=======================================//
	//          Transfer values....          //
	//=======================================//

	
	size_t bytes = N * sizeof(float); // Size, in bytes, of each vector

	float* atoms_x_h; // x vector (host)
	check(hipHostMalloc(&atoms_x_h, bytes));

	float* atoms_y_h; // y vector (host)
	check(hipHostMalloc(&atoms_y_h, bytes));

	Eigen::ArrayXf::Map(atoms_x_h, atoms_x.rows()) = atoms_x; // Map to x vector (host)
	Eigen::ArrayXf::Map(atoms_y_h, atoms_y.rows()) = atoms_y; // Map to y vector (host)

	float* atoms_x_d; // x vector (device)
	check(hipMalloc(&atoms_x_d, bytes));

	float* atoms_y_d; // y vector (device)
	check(hipMalloc(&atoms_y_d, bytes));

	check(hipMemcpy(atoms_x_d, atoms_x_h, bytes, hipMemcpyHostToDevice));
	check(hipMemcpy(atoms_y_d, atoms_y_h, bytes, hipMemcpyHostToDevice));

	//======================================================//
	// Matrix grids
	int n_blocks(div_up(N, sqrt(threads_per_block)));

	dim3 blockSize = dim3(sqrt(threads_per_block), sqrt(threads_per_block));
	dim3 gridSize = dim3(n_blocks, n_blocks);

	//======================================================//
	// Pairwise distance and difference calculation
	float* diffs_x_d;
	check(hipMalloc(&diffs_x_d, N * bytes));

	float* diffs_y_d;
	check(hipMalloc(&diffs_y_d, N * bytes));

	calc_force_m << <gridSize, blockSize >> > (diffs_x_d, diffs_y_d, atoms_x_d, atoms_y_d, SD_factor, N);
	//======================================================//
	// Force calculation (rowise matrix reduction by CUBLAS)

	float* force_x_d; // Force x vector (device)
	check(hipMalloc(&force_x_d, bytes));

	float* force_y_d; // Force y vector (device)
	check(hipMalloc(&force_y_d, bytes));

	float *d_ones; // vector of ones to multiply matrix with (device)
	check(hipMalloc((void **)&d_ones, bytes));
	const float value = 1.f;
	initKernel << <n_blocks, threads_per_block >> > (d_ones, value, N); // initialize vector with ones using CUDA

	hipblasHandle_t handle;
	cublascheck(hipblasCreate(&handle)); // construct cublas handle

	float alpha = 1.f;
	float beta = 0.f;
	cublascheck(hipblasSgemv(handle, HIPBLAS_OP_T, N, N, &alpha, thrust::raw_pointer_cast(diffs_x_d), N,
		thrust::raw_pointer_cast(d_ones), 1, &beta, thrust::raw_pointer_cast(force_x_d), 1)); // rowwise multiplication x
	cublascheck(hipblasSgemv(handle, HIPBLAS_OP_T, N, N, &alpha, thrust::raw_pointer_cast(diffs_y_d), N,
		thrust::raw_pointer_cast(d_ones), 1, &beta, thrust::raw_pointer_cast(force_y_d), 1)); // rowwise multiplication y

	cublascheck(hipblasDestroy(handle)); // destroy cublas handle to avoid malloc errors
	//======================================================//

	//=======================================//
	//          Retrieve values....          //
	//=======================================//

	check(hipPeekAtLastError());
	check(hipDeviceSynchronize());

	float* force_x_h; // Force x vector (device)
	check(hipHostMalloc(&force_x_h, bytes));

	float* force_y_h; // Force y vector (device)
	check(hipHostMalloc(&force_y_h, bytes));

	check(hipMemcpy(force_x_h, force_x_d, bytes, hipMemcpyDeviceToHost)); // Copy forces x to device 
	check(hipMemcpy(force_y_h, force_y_d, bytes, hipMemcpyDeviceToHost)); // Copy forces y to device

	*force_x = Eigen::Map<Eigen::ArrayXf>(force_x_h, N); // Map forces x to Eigen array 
	*force_y = Eigen::Map<Eigen::ArrayXf>(force_y_h, N); // Map forces y to Eigen array

	/* Destroy all memory allocation pointers and free memory */
	check(hipFree(atoms_x_d));
	check(hipHostFree(atoms_x_h));
	check(hipFree(atoms_y_d));
	check(hipHostFree(atoms_y_h));

	check(hipFree(diffs_x_d));
	check(hipFree(diffs_y_d));

	check(hipFree(force_x_d));
	check(hipHostFree(force_x_h));
	check(hipFree(force_y_d));
	check(hipHostFree(force_y_h));
}

/*-----------------------------------------------------------*/
/*              Repulsive force evaluation (GPU)             */
/*-----------------------------------------------------------*/
DLL_API void tracker_gpu(Eigen::ArrayXXf *G, Eigen::ArrayXf *p, Eigen::ArrayXf atoms_x, Eigen::ArrayXf atoms_y,
	const int n_pop, const int n_grids, const int threads_per_block)
{

	const int N_rows(n_pop); // number of rows
	const int N_cols(n_grids * n_grids); // number of rows

	//=======================================//
	//          Transfer values....          //
	//=======================================//

	//======================================================//
	// Copy position vectors to device
	size_t bytes_rows = N_rows * sizeof(float); // Size, in bytes, of each vector
	size_t bytes_cols = N_cols * sizeof(float); // Size, in bytes, of each vector

	float* atoms_x_h; // x vector (host)
	check(hipHostMalloc(&atoms_x_h, bytes_rows));

	float* atoms_y_h; // y vector (host)
	check(hipHostMalloc(&atoms_y_h, bytes_rows));

	Eigen::ArrayXf::Map(atoms_x_h, atoms_x.rows()) = atoms_x; // Map to x vector (host)
	Eigen::ArrayXf::Map(atoms_y_h, atoms_y.rows()) = atoms_y; // Map to y vector (host)

	float* atoms_x_d; // x vector (device)
	check(hipMalloc(&atoms_x_d, bytes_rows));

	float* atoms_y_d; // y vector (device)
	check(hipMalloc(&atoms_y_d, bytes_rows));

	check(hipMemcpy(atoms_x_d, atoms_x_h, bytes_rows, hipMemcpyHostToDevice));
	check(hipMemcpy(atoms_y_d, atoms_y_h, bytes_rows, hipMemcpyHostToDevice));

	//======================================================//
	// Copy ground covered matrix to device
	float* G_h; // Force x vector (device)
	check(hipHostMalloc(&G_h, N_rows * N_cols * sizeof(float)));
	Eigen::Map<Eigen::Array<float, Eigen::Dynamic, Eigen::Dynamic, Eigen::RowMajor > >(G_h, N_rows, N_cols) = *G;

	//for (int i(0); i < N_rows; ++i) {
	//	for (int j(0); j < N_cols; ++j) {
	//		//cout << "(" << i << "," << j << "): " << G_h[i + N_cols * j] << ", ";
	//		cout << G_h[j + N_cols * i] << ", ";
	//	}
	//	cout << endl;
	//}
	//cout << endl;

	float* G_d;
	check(hipMalloc(&G_d, N_rows * N_cols * sizeof(float)));

	check(hipMemcpy(G_d, G_h, N_rows * N_cols * sizeof(float), hipMemcpyHostToDevice)); // Copy tracking matrix to device 

	//======================================================//
	// Matrix grids
	float h_t = 2 * floor((sqrt(1024 * ((float)N_rows / (float)N_cols))) / 2); // block height in threads (floor up to nearest even number)
	float w_t = 2 * floor((h_t * ((float)N_cols / (float)N_rows)) / 2); // block width in threads (floor up to nearest even number)

	int n_blocks_h(div_up(N_cols, w_t));
	int n_blocks_w(div_up(N_rows, h_t));

	dim3 blockSize = dim3(h_t, w_t);
	dim3 gridSize = dim3(n_blocks_h, n_blocks_w);
	//dim3 blockSize = dim3(1, 1);
	//dim3 gridSize = dim3(10, 9);

	//======================================================//
	// Pairwise distance and difference calculation

	calc_tracking_matrix << <gridSize, blockSize >> > (G_d, atoms_x_d, atoms_y_d, n_pop, n_grids, N_rows, N_cols);

	//======================================================//
	// Percentage covered (rowise matrix reduction by CUBLAS)

	float* p_d; // percentage vector (device)
	check(hipMalloc(&p_d, bytes_rows));

	float *d_ones; // vector of ones to multiply matrix with (device)
	check(hipMalloc((void **)&d_ones, bytes_cols));
	const float value = 1.f;

	int n_blocks(div_up(N_cols, sqrt(threads_per_block)));
	initKernel << <n_blocks, threads_per_block >> > (d_ones, value, N_cols); // initialize vector with ones using CUDA

	hipblasHandle_t handle;
	cublascheck(hipblasCreate(&handle)); // construct cublas handle

	float alpha = 1.f;
	float beta = 0.f;
	cublascheck(hipblasSgemv(handle, HIPBLAS_OP_T, N_cols, N_rows, &alpha, thrust::raw_pointer_cast(G_d), N_cols,
		thrust::raw_pointer_cast(d_ones), 1, &beta, thrust::raw_pointer_cast(p_d), 1)); // rowwise multiplication

	cublascheck(hipblasDestroy(handle)); // destroy cublas handle to avoid malloc errors

	//=======================================//
	//          Retrieve values....          //
	//=======================================//

	//======================================================//
	// Copy percentage covered vector to host
	float* p_h; // percentage vector (device)
	check(hipHostMalloc(&p_h, bytes_rows));

	check(hipMemcpy(p_h, p_d, bytes_rows, hipMemcpyDeviceToHost)); // Copy percentage to device 

	*p = Eigen::Map<Eigen::ArrayXf>(p_h, N_rows); // Map percentage to Eigen array 

	//======================================================//
	// Copy ground covered matrix to host
	check(hipPeekAtLastError());
	check(hipDeviceSynchronize());

	check(hipMemcpy(G_h, G_d, N_rows * N_cols * sizeof(float), hipMemcpyDeviceToHost)); // Copy tracking matrix to device 

	//for (int i(0); i < N_rows; ++i) {
	//	for (int j(0); j < N_cols; ++j) {
	//		//cout << "(" << i << "," << j << "): " << G_h[i + N_cols * j] << ", ";
	//		cout << G_h[j + N_cols * i] << ", ";
	//	}
	//	cout << endl;
	//}
	//cout << endl;

	// Map rowwise format
	*G = Eigen::Map<Eigen::Array<float, Eigen::Dynamic, Eigen::Dynamic, Eigen::RowMajor > >(G_h, N_rows, N_cols);

	//for (int i(0); i < N_rows; ++i) {
	//	for (int j(0); j < N_cols; ++j) {
	//		//cout << "(" << i << "," << j << "): " << G->block(i,j,1,1) << ", ";
	//		cout << G->block(i, j, 1, 1) << ", ";
	//	}
	//	cout << endl;
	//}

	/* Destroy all memory allocation pointers and free memory */
	check(hipFree(atoms_x_d));
	check(hipHostFree(atoms_x_h));
	check(hipFree(atoms_y_d));
	check(hipHostFree(atoms_y_h));

	check(hipFree(G_d));
	check(hipHostFree(G_h));

	check(hipFree(p_d));
	check(hipHostFree(p_h));
}
